#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda_bfs.cuh"
#include<stdio.h>
#define BLOCK_SIZE 256

// Device pointers to be shared across functions
int *d_edges, *d_offsets, *d_visited, *d_output;

// CUDA Initialization Function
void cuda_init(int m, int n, const std::vector<int> &edges, const std::vector<int> &offsets) {
    hipMalloc(&d_edges, m * sizeof(int));
    hipMalloc(&d_offsets, (n + 1) * sizeof(int));
    hipMalloc(&d_visited, (n + 1) * sizeof(int)); // Extra slot for traversal counter
    hipMalloc(&d_output, n * sizeof(int));

    hipMemcpy(d_edges, edges.data(), m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, offsets.data(), (n + 1) * sizeof(int), hipMemcpyHostToDevice);
}

// CUDA Cleanup Function
void cuda_cleanup() {
    hipFree(d_edges);
    hipFree(d_offsets);
    hipFree(d_visited);
    hipFree(d_output);
}

// BFS Kernel
__global__ void bfs_kernel(int *edges, int *offsets, int *visited, int *frontier, int frontier_size, int *next_frontier, int *next_frontier_size, int num_vertices, int *output) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= frontier_size) return;

    int current_vertex = frontier[tid];
    for (int i = offsets[current_vertex]; i < offsets[current_vertex + 1]; i++) {
        int neighbor = edges[i];

        // Only add unvisited neighbors to the next frontier
        if (atomicExch(&visited[neighbor], 1) == 0) {
            int idx = atomicAdd(next_frontier_size, 1);
            next_frontier[idx] = neighbor;

            // Add to the output array
            int output_idx = atomicAdd(&visited[num_vertices], 1); // Use visited[num_vertices] as a counter
            output[output_idx] = neighbor;

            // Debugging print
            // printf("Adding vertex %d to output at index %d\n", neighbor + 1, output_idx);
        }
    }
}

// BFS Function
void cuda_bfs(int num_vertices, int start_vertex, std::vector<int> &output) {
    int *frontier, *next_frontier, *d_output_size;
    hipMalloc(&frontier, num_vertices * sizeof(int));
    hipMalloc(&next_frontier, num_vertices * sizeof(int));
    hipMalloc(&d_output_size, sizeof(int));

    // Initialize visited and output arrays
    hipMemset(d_visited, 0, (num_vertices + 1) * sizeof(int)); // Extra slot for traversal counter
    hipMemset(d_output, -1, num_vertices * sizeof(int));
    hipMemset(d_output_size, 0, sizeof(int));

    // Add the start vertex to the frontier and mark it visited
    int frontier_size = 1;
    hipMemcpy(frontier, &start_vertex, sizeof(int), hipMemcpyHostToDevice);

    // Host array for visited
    int *h_visited = new int[num_vertices + 1](); // +1 for traversal counter
    h_visited[start_vertex] = 1;                 // Mark the start vertex as visited
    h_visited[num_vertices] = 1;                 // Initialize traversal counter to 1
    hipMemcpy(d_visited, h_visited, (num_vertices + 1) * sizeof(int), hipMemcpyHostToDevice);

    // Host array for output
    int *h_output = new int[num_vertices];       // Host array for output
    std::fill(h_output, h_output + num_vertices, -1); // Initialize to -1
    h_output[0] = start_vertex;                  // Add the start vertex at the beginning
    hipMemcpy(d_output, h_output, num_vertices * sizeof(int), hipMemcpyHostToDevice);

    // Perform BFS
    while (frontier_size > 0) {
        hipMemset(d_output_size, 0, sizeof(int));

        bfs_kernel<<<(frontier_size + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
            d_edges, d_offsets, d_visited, frontier, frontier_size, next_frontier, d_output_size, num_vertices, d_output);
        hipDeviceSynchronize();

        hipMemcpy(&frontier_size, d_output_size, sizeof(int), hipMemcpyDeviceToHost);

        if (frontier_size > 0) {
            hipMemcpy(frontier, next_frontier, frontier_size * sizeof(int), hipMemcpyDeviceToDevice);
        }
    }

    // Copy results back to host
    output.resize(num_vertices);
    hipMemcpy(output.data(), d_output, num_vertices * sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup host memory
    delete[] h_visited;
    delete[] h_output;

    // Free allocated device memory
    hipFree(frontier);
    hipFree(next_frontier);
    hipFree(d_output_size);
}